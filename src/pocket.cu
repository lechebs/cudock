#include "pocket.hpp"

#include <iostream>

#include <hip/hip_runtime_api.h>

#include "utils.cuh"

namespace
{
    void _alloc_global(float *src[], float *dst[], int size, int num_buffers)
    {
        for (int i = 0; i < num_buffers; ++i) {
            CUDA_CHECK_ERR(hipMalloc(&dst[i], sizeof(float) * size));
            CUDA_CHECK_ERR(hipMemcpy(dst[i],
                                      src[i],
                                      sizeof(float) * size,
                                      hipMemcpyHostToDevice));
        }
    }

    void _free_global(float *buffers[], int num_buffers)
    {
        for (int i = 0; i < num_buffers; ++i) {
            CUDA_CHECK_ERR(hipFree(buffers[i]));
        }
    }

    void _alloc_textures(float *src[],
                         hipArray_t dst_arrays[],
                         hipTextureObject_t dst_textures[],
                         int num_textures,
                         int width,
                         int height,
                         int depth)
    {
        hipResourceDesc res_desc = {};
        res_desc.resType = hipResourceTypeArray;
        hipChannelFormatDesc fmt_desc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        hipExtent extent = make_hipExtent(width, height, depth);

        hipTextureDesc tex_desc = {};
        tex_desc.addressMode[0] = hipAddressModeClamp;
        tex_desc.addressMode[1] = hipAddressModeClamp;
        tex_desc.addressMode[2] = hipAddressModeClamp;
        //tex_desc.normalizedCoords = false;
        tex_desc.filterMode = hipFilterModePoint;
        //tex_desc.disableTrilinearOptimization = true;

        hipResourceViewDesc view_desc = {};
        view_desc.format = hipResViewFormatFloat1;
        view_desc.width = width;
        view_desc.height = height;
        view_desc.depth = depth;

        for (int i = 0; i < num_textures; ++i) {
            CUDA_CHECK_ERR(hipMalloc3DArray(&dst_arrays[i], &fmt_desc, extent));

            hipMemcpy3DParms parms = {0};
            parms.srcPtr = make_hipPitchedPtr(src[i],
                                               width * sizeof(float),
                                               width,
                                               height);
            parms.dstArray = dst_arrays[i];
            parms.extent = extent;
            parms.kind = hipMemcpyHostToDevice;
            CUDA_CHECK_ERR(hipMemcpy3D(&parms));

            res_desc.res.array.array = dst_arrays[i];
            CUDA_CHECK_ERR(hipCreateTextureObject(&dst_textures[i],
                                                   &res_desc,
                                                   &tex_desc,
                                                   &view_desc));
        }
    }

    void _free_textures(hipArray_t arrays[],
                        hipTextureObject_t textures[],
                        int num_textures)
    {
        for (int i = 0; i < num_textures; ++i) {
            CUDA_CHECK_ERR(hipFreeArray(arrays[i]));
            CUDA_CHECK_ERR(hipDestroyTextureObject(textures[i]));
        }
    }
};

namespace cuDock
{
    bool Pocket::is_on_gpu() const
    {
        return _is_on_gpu[GPU_GMEM] || _is_on_gpu[GPU_TMEM];
    }

    bool Pocket::is_on_gpu(enum GPUMemType mem_type) const
    {
        return _is_on_gpu[mem_type];
    }

    void Pocket::to_gpu(enum GPUMemType mem_type)
    {
        if (!_is_on_gpu[mem_type]) {
            _is_on_gpu[mem_type] = true;

            if (mem_type == GPU_GMEM) {
                _alloc_global(_voxels.data(),
                              _gpu_global_voxels.data(),
                              get_size(),
                              NUM_CHANNELS);
            } else if (mem_type == GPU_TMEM) {
                _alloc_textures(_voxels.data(),
                                _gpu_array_voxels.data(),
                                _gpu_texture_voxels.data(),
                                NUM_CHANNELS,
                                get_shape(0),
                                get_shape(1),
                                get_shape(2));
            }
        }

    }

    void Pocket::off_gpu(enum GPUMemType mem_type)
    {
        if (_is_on_gpu[mem_type]) {
            _is_on_gpu[mem_type] = false;

            if (mem_type == GPU_GMEM) {
                _free_global(_gpu_global_voxels.data(), NUM_CHANNELS);
            } else if (mem_type == GPU_TMEM) {
                _free_textures(_gpu_array_voxels.data(),
                               _gpu_texture_voxels.data(),
                               NUM_CHANNELS);
            }
        }
    }

    const std::array<float *, Pocket::NUM_CHANNELS>
    &Pocket::get_gpu_gmem_voxels() const
    {
        return _gpu_global_voxels;
    }

    const std::array<hipTextureObject_t, Pocket::NUM_CHANNELS>
    &Pocket::get_gpu_tmem_voxels() const
    {
        return _gpu_texture_voxels;
    }
};
