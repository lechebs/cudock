#include "pocket.hpp"

#include <iostream>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "utils.cuh"
#include "swizzling.cuh"
#include "docker.hpp"

namespace
{
    void _alloc_global(float *src[], float *dst[], int size, int num_buffers)
    {
        for (int i = 0; i < num_buffers; ++i) {
            CUDA_CHECK_ERR(hipMalloc(&dst[i], sizeof(float) * size));
            CUDA_CHECK_ERR(hipMemcpy(dst[i],
                                      src[i],
                                      sizeof(float) * size,
                                      hipMemcpyHostToDevice));
        }
    }

    void _free_global(float *buffers[], int num_buffers)
    {
        for (int i = 0; i < num_buffers; ++i) {
            CUDA_CHECK_ERR(hipFree(buffers[i]));
        }
    }

    void _alloc_textures(float *src[],
                         hipArray_t dst_arrays[],
                         hipTextureObject_t dst_textures[],
                         int num_textures,
                         int width,
                         int height,
                         int depth,
                         bool lerp)
    {
        hipResourceDesc res_desc = {};
        res_desc.resType = hipResourceTypeArray;
        hipChannelFormatDesc fmt_desc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        hipExtent extent = make_hipExtent(width, height, depth);

        hipTextureDesc tex_desc = {};
        tex_desc.addressMode[0] = hipAddressModeClamp;
        tex_desc.addressMode[1] = hipAddressModeClamp;
        tex_desc.addressMode[2] = hipAddressModeClamp;
        //tex_desc.normalizedCoords = false;
        tex_desc.filterMode = lerp ? hipFilterModeLinear :
                                     hipFilterModePoint;
        //tex_desc.disableTrilinearOptimization = true;

        hipResourceViewDesc view_desc = {};
        view_desc.format = hipResViewFormatFloat1;
        view_desc.width = width;
        view_desc.height = height;
        view_desc.depth = depth;

        for (int i = 0; i < num_textures; ++i) {
            CUDA_CHECK_ERR(hipMalloc3DArray(&dst_arrays[i], &fmt_desc, extent));

            hipMemcpy3DParms parms = {0};
            parms.srcPtr = make_hipPitchedPtr(src[i],
                                               width * sizeof(float),
                                               width,
                                               height);
            parms.dstArray = dst_arrays[i];
            parms.extent = extent;
            parms.kind = hipMemcpyHostToDevice;
            CUDA_CHECK_ERR(hipMemcpy3D(&parms));

            res_desc.res.array.array = dst_arrays[i];
            CUDA_CHECK_ERR(hipCreateTextureObject(&dst_textures[i],
                                                   &res_desc,
                                                   &tex_desc,
                                                   &view_desc));
        }
    }

    void _free_textures(hipArray_t arrays[],
                        hipTextureObject_t textures[],
                        int num_textures)
    {
        for (int i = 0; i < num_textures; ++i) {
            CUDA_CHECK_ERR(hipFreeArray(arrays[i]));
            CUDA_CHECK_ERR(hipDestroyTextureObject(textures[i]));
        }
    }
};

namespace cuDock
{
    bool Pocket::is_on_gpu() const
    {
        return _is_on_gpu[GPU_GMEM] ||
               _is_on_gpu[GPU_GMEM_SWIZZLED] ||
               _is_on_gpu[GPU_TMEM];
    }

    bool Pocket::is_on_gpu(enum GPUMemType mem_type) const
    {
        return _is_on_gpu[mem_type];
    }

    void Pocket::to_gpu(enum GPUMemType mem_type)
    {
        if (!_is_on_gpu[mem_type]) {
            _is_on_gpu[mem_type] = true;

            if (mem_type == GPU_GMEM) {
                _alloc_global(_voxels.data(),
                              _gpu_global_voxels.data(),
                              get_size(),
                              NUM_CHANNELS);

            } else if (mem_type == GPU_GMEM_SWIZZLED) {

                int w = get_shape(0);
                int h = get_shape(1);
                int d = get_shape(2);

                // TODO: make parameter
                int tile_size_in_bits = 4;
                int swizzled_size =
                    Swizzling::get_swizzled_size(w, h, d, tile_size_in_bits);

                // Testing compressible memory
                /*
                for (int c = 0; c < NUM_CHANNELS; ++c) {

                hipMemAllocationProp prop = {};
                prop.type = hipMemAllocationTypePinned;
                prop.location.type = hipMemLocationTypeDevice;
                prop.location.id = 0;
                prop.allocFlags.compressionType =
                    CU_MEM_ALLOCATION_COMP_GENERIC;

                size_t granularity = 0;
                hipError_t res = hipMemGetAllocationGranularity(
                    &granularity,
                    &prop,
                    hipMemAllocationGranularityMinimum);
                std::cout << "hipMemGetAllocationGranularity: " << res << std::endl;

                size_t size = sizeof(float) * swizzled_size;
                size_t padded_size = ((size - 1) / granularity + 1) *
                                     granularity;

                hipMemGenericAllocationHandle_t allocHandle;
                res = hipMemCreate(&allocHandle, padded_size, &prop, 0);
                std::cout << "hipMemCreate: " << res << std::endl;

                hipMemGetAllocationPropertiesFromHandle(&prop,
                                                       allocHandle);

                if (prop.allocFlags.compressionType ==
                    CU_MEM_ALLOCATION_COMP_GENERIC)
                {
                    std::cout << "Obtained compressible memory" << std::endl;
                }

                hipDeviceptr_t ptr;
                res = hipMemAddressReserve(&ptr, padded_size, 0, 0, 0);
                std::cout << "hipMemAddressReserve: " << res << std::endl;

                res = hipMemMap(ptr, padded_size, 0, allocHandle, 0);
                std::cout << "hipMemMap: " << res << std::endl;

                // Make the address accessible
                hipMemAccessDesc accessDesc = {};
                accessDesc.location.type = hipMemLocationTypeDevice;
                accessDesc.location.id = 0;
                accessDesc.flags = hipMemAccessFlagsProtReadWrite;

                hipMemSetAccess(ptr, padded_size, &accessDesc, 1);
                _gpu_global_voxels[c] = (float *) ptr;
                }
                    */

                std::array<float *, NUM_CHANNELS> voxels_swizzled;
                for (int c = 0; c < NUM_CHANNELS; ++c) {
                    voxels_swizzled[c] = new float[swizzled_size];
                    Swizzling::
                    to_swizzled_format(w,
                                       h,
                                       d,
                                       tile_size_in_bits,
                                       _voxels[c],
                                       voxels_swizzled[c]);
                }

                _alloc_global(voxels_swizzled.data(),
                              _gpu_global_voxels.data(),
                              swizzled_size,
                              NUM_CHANNELS);

                for (int c = 0; c < NUM_CHANNELS; ++c) {
                    delete[] voxels_swizzled[c];
                }

            } else if (mem_type == GPU_TMEM) {
                _alloc_textures(_voxels.data(),
                                _gpu_array_voxels.data(),
                                _gpu_texture_voxels.data(),
                                NUM_CHANNELS,
                                get_shape(0),
                                get_shape(1),
                                get_shape(2),
                                get_interpolate() == LIN_INTERPOLATE);
            }
        }

    }

    void Pocket::off_gpu(enum GPUMemType mem_type)
    {
        if (_is_on_gpu[mem_type]) {
            _is_on_gpu[mem_type] = false;

            if (mem_type == GPU_GMEM|| mem_type == GPU_GMEM_SWIZZLED) {
                _free_global(_gpu_global_voxels.data(), NUM_CHANNELS);
            } else if (mem_type == GPU_TMEM) {
                _free_textures(_gpu_array_voxels.data(),
                               _gpu_texture_voxels.data(),
                               NUM_CHANNELS);
            }
        }
    }

    const std::array<float *, Pocket::NUM_CHANNELS>
    &Pocket::get_gpu_gmem_voxels() const
    {
        return _gpu_global_voxels;
    }

    const std::array<hipTextureObject_t, Pocket::NUM_CHANNELS>
    &Pocket::get_gpu_tmem_voxels() const
    {
        return _gpu_texture_voxels;
    }
};
