#include "pocket.hpp"

#include <iostream>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>

#include "utils.cuh"
#include "swizzling.cuh"
#include "docker.hpp"

namespace
{
    void _alloc_global(float *src[], float *dst[], int size, int num_buffers)
    {
        for (int i = 0; i < num_buffers; ++i) {
            CUDA_CHECK_ERR(hipMalloc(&dst[i], sizeof(float) * size));
            CUDA_CHECK_ERR(hipMemcpy(dst[i],
                                      src[i],
                                      sizeof(float) * size,
                                      hipMemcpyHostToDevice));
        }
    }

    void _free_global(float *buffers[], int num_buffers)
    {
        for (int i = 0; i < num_buffers; ++i) {
            CUDA_CHECK_ERR(hipFree(buffers[i]));
        }
    }

    void _alloc_textures(float *src[],
                         hipArray_t dst_arrays[],
                         hipTextureObject_t dst_textures[],
                         int num_textures,
                         int width,
                         int height,
                         int depth)
    {
        hipResourceDesc res_desc = {};
        res_desc.resType = hipResourceTypeArray;
        hipChannelFormatDesc fmt_desc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        hipExtent extent = make_hipExtent(width, height, depth);

        hipTextureDesc tex_desc = {};
        tex_desc.addressMode[0] = hipAddressModeClamp;
        tex_desc.addressMode[1] = hipAddressModeClamp;
        tex_desc.addressMode[2] = hipAddressModeClamp;
        //tex_desc.normalizedCoords = false;
        tex_desc.filterMode = hipFilterModePoint;
        //tex_desc.disableTrilinearOptimization = true;

        hipResourceViewDesc view_desc = {};
        view_desc.format = hipResViewFormatFloat1;
        view_desc.width = width;
        view_desc.height = height;
        view_desc.depth = depth;

        for (int i = 0; i < num_textures; ++i) {
            CUDA_CHECK_ERR(hipMalloc3DArray(&dst_arrays[i], &fmt_desc, extent));

            hipMemcpy3DParms parms = {0};
            parms.srcPtr = make_hipPitchedPtr(src[i],
                                               width * sizeof(float),
                                               width,
                                               height);
            parms.dstArray = dst_arrays[i];
            parms.extent = extent;
            parms.kind = hipMemcpyHostToDevice;
            CUDA_CHECK_ERR(hipMemcpy3D(&parms));

            res_desc.res.array.array = dst_arrays[i];
            CUDA_CHECK_ERR(hipCreateTextureObject(&dst_textures[i],
                                                   &res_desc,
                                                   &tex_desc,
                                                   &view_desc));
        }
    }

    void _free_textures(hipArray_t arrays[],
                        hipTextureObject_t textures[],
                        int num_textures)
    {
        for (int i = 0; i < num_textures; ++i) {
            CUDA_CHECK_ERR(hipFreeArray(arrays[i]));
            CUDA_CHECK_ERR(hipDestroyTextureObject(textures[i]));
        }
    }
};

namespace cuDock
{
    bool Pocket::is_on_gpu() const
    {
        return _is_on_gpu[GPU_GMEM] ||
               _is_on_gpu[GPU_GMEM_SWIZZLED] ||
               _is_on_gpu[GPU_TMEM];
    }

    bool Pocket::is_on_gpu(enum GPUMemType mem_type) const
    {
        return _is_on_gpu[mem_type];
    }

    void Pocket::to_gpu(enum GPUMemType mem_type)
    {
        if (!_is_on_gpu[mem_type]) {
            _is_on_gpu[mem_type] = true;

            if (mem_type == GPU_GMEM) {
                _alloc_global(_voxels.data(),
                              _gpu_global_voxels.data(),
                              get_size(),
                              NUM_CHANNELS);

            } else if (mem_type == GPU_GMEM_SWIZZLED) {

                int w = get_shape(0);
                int h = get_shape(1);
                int d = get_shape(2);

                int tile_size_in_bits = 5;
                int swizzled_size =
                    Swizzling::get_swizzled_size(w, h, d, tile_size_in_bits);

                std::array<float *, NUM_CHANNELS> voxels_swizzled;
                for (int c = 0; c < NUM_CHANNELS; ++c) {
                    voxels_swizzled[c] = new float[swizzled_size];
                    Swizzling::
                    to_swizzled_format(w,
                                       h,
                                       d,
                                       tile_size_in_bits,
                                       _voxels[c],
                                       voxels_swizzled[c]);
                }

                _alloc_global(voxels_swizzled.data(),
                              _gpu_global_voxels.data(),
                              swizzled_size,
                              NUM_CHANNELS);

                for (int c = 0; c < NUM_CHANNELS; ++c) {
                    delete[] voxels_swizzled[c];
                }

            } else if (mem_type == GPU_TMEM) {
                _alloc_textures(_voxels.data(),
                                _gpu_array_voxels.data(),
                                _gpu_texture_voxels.data(),
                                NUM_CHANNELS,
                                get_shape(0),
                                get_shape(1),
                                get_shape(2));
            }
        }

    }

    void Pocket::off_gpu(enum GPUMemType mem_type)
    {
        if (_is_on_gpu[mem_type]) {
            _is_on_gpu[mem_type] = false;

            if (mem_type == GPU_GMEM|| mem_type == GPU_GMEM_SWIZZLED) {
                _free_global(_gpu_global_voxels.data(), NUM_CHANNELS);
            } else if (mem_type == GPU_TMEM) {
                _free_textures(_gpu_array_voxels.data(),
                               _gpu_texture_voxels.data(),
                               NUM_CHANNELS);
            }
        }
    }

    const std::array<float *, Pocket::NUM_CHANNELS>
    &Pocket::get_gpu_gmem_voxels() const
    {
        return _gpu_global_voxels;
    }

    const std::array<hipTextureObject_t, Pocket::NUM_CHANNELS>
    &Pocket::get_gpu_tmem_voxels() const
    {
        return _gpu_texture_voxels;
    }
};
