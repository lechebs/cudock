#include "hip/hip_runtime.h"
#include "docker.hpp"

#include <iostream>
#include <vector>

#include "ligand.hpp"
#include "utils.cuh"

#define WARP_SIZE 32
#define BLOCK_SIZE 128

namespace
{
    __constant__ struct cuDock::Ligand::GPUData _gpu_ligand_data;

    __device__ void _compute_rot_mat(float3 r, float mat[])
    {
        float sinrx = __sinf(r.x);
        float cosrx = __cosf(r.x);
        float sinry = __sinf(r.y);
        float cosry = __cosf(r.x);
        float sinrz = __sinf(r.z);
        float cosrz = __cosf(r.z);

        mat[0] = cosry * cosrz;
        mat[1] = sinrx * sinry * cosrz - cosrx * sinrz;
        mat[2] = cosrx * sinry * cosrz + sinrx * sinrz;

        mat[3] = cosry * sinrz;
        mat[4] = sinrx * sinry * sinrz + cosrx * sinrz;
        mat[5] = cosrx * sinry * sinrz - sinrx * cosrz;

        mat[6] = -sinry;
        mat[7] = sinrx * cosry;
        mat[8] = cosrx * cosry;
    }

    __device__ __inline__ void
    _transform_atom_pos(float x,
                        float y,
                        float z,
                        float3 t,
                        const float r_mat[],
                        float3 &dst)
    {
        dst.x = r_mat[0] * x + r_mat[1] * y + r_mat[2] * z + t.x;
        dst.y = r_mat[3] * x + r_mat[4] * y + r_mat[5] * z + t.y;
        dst.z = r_mat[6] * x + r_mat[7] * y + r_mat[8] * z + t.z;
    }

    __global__ void _score_gmem(const float * const voxels[],
                                int num_channels,
                                const float3 *translations,
                                const float3 *rotations,
                                int num_poses,
                                int num_atoms,
                                int block_size)
    {
        // Broadcasting to warp
        float3 t = translations[blockIdx.x];
        float3 r = rotations[blockIdx.x];

        float r_mat[9];
        _compute_rot_mat(r, r_mat);

        for (int i = 0; i < num_atoms; i += block_size) {
            float x = _gpu_ligand_data.atoms_x[threadIdx.x];
            float y = _gpu_ligand_data.atoms_y[threadIdx.x];
            float z = _gpu_ligand_data.atoms_z[threadIdx.x];

            float3 pos;
            _transform_atom_pos(x, y, z, t, r_mat, pos);
        }
    }

    __global__ void _score_tmem(const hipTextureObject_t textures[],
                                int num_textures,
                                const float3 *translations,
                                const float3 *rotations,
                                int num_poses,
                                int num_atoms,
                                int block_size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx > num_poses) {
            return;
        }
    }
}

namespace cuDock
{
    void Docker::to_gpu()
    {
        int num_poses = _translations.size();
        if (num_poses == 0) {
            return;
        }

        int alloc_size = num_poses * sizeof(float3);

        if (!_is_on_gpu) {
            _is_on_gpu = true;
            CUDA_CHECK_ERR(hipMalloc(&_gpu_translations, alloc_size));
            CUDA_CHECK_ERR(hipMalloc(&_gpu_rotations, alloc_size));
        }

        std::vector<float3> t_buffer;
        std::vector<float3> r_buffer;
        t_buffer.reserve(num_poses);
        r_buffer.reserve(num_poses);

        for (int i = 0; i < num_poses; ++i) {
            t_buffer[i].x = _translations[i][0];
            t_buffer[i].y = _translations[i][1];
            t_buffer[i].z = _translations[i][2];
            r_buffer[i].x = _rotations[i][0];
            r_buffer[i].y = _rotations[i][1];
            r_buffer[i].z = _rotations[i][2];
        }

        // Copying poses to global memory

        CUDA_CHECK_ERR(hipMemcpy(_gpu_translations,
                                  t_buffer.data(),
                                  alloc_size,
                                  hipMemcpyHostToDevice));

        CUDA_CHECK_ERR(hipMemcpy(_gpu_rotations,
                                  r_buffer.data(),
                                  alloc_size,
                                  hipMemcpyHostToDevice));

        // Copying ligand data to SoA constant memory buffer

        const std::vector<Ligand::Atom> &atoms = _ligand.get_atoms();
        int num_atoms = atoms.size();

        struct Ligand::GPUData ligand_data;
        for (int i = 0; i < num_atoms; ++i) {
            const Ligand::Atom &atom = atoms[i];
            ligand_data.atoms_x[i] = atom.pos[0];
            ligand_data.atoms_y[i] = atom.pos[1];
            ligand_data.atoms_z[i] = atom.pos[2];
            ligand_data.atom_type[i] = atom.type;
        }

        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(_gpu_ligand_data),
                                          &ligand_data,
                                          sizeof(struct Ligand::GPUData)));
    }

    void Docker::off_gpu()
    {
        if (_is_on_gpu) {
            _is_on_gpu = false;
            CUDA_CHECK_ERR(hipFree(_gpu_translations));
            CUDA_CHECK_ERR(hipFree(_gpu_rotations));
        }
    }

    void Docker::_score_poses_gpu(int num_poses)
    {
        int num_atoms = _ligand.get_num_atoms();
        int block_size = num_atoms / WARP_SIZE * WARP_SIZE;

        if (_pocket.is_on_gpu(GPU_GMEM)) {
            CUDA_TIME_EXEC("_score_gmem", [&](){
                _score_gmem<<<
                    num_poses,
                    block_size>>>(_pocket.get_gpu_gmem_voxels().data(),
                                  Pocket::NUM_CHANNELS,
                                  _gpu_translations,
                                  _gpu_rotations,
                                  num_poses,
                                  num_atoms,
                                  block_size);
            });
        } else {
            CUDA_TIME_EXEC("_score_tmem", [&](){
                _score_tmem<<<
                    num_poses,
                    block_size>>>(_pocket.get_gpu_tmem_voxels().data(),
                                  Pocket::NUM_CHANNELS,
                                  _gpu_translations,
                                  _gpu_rotations,
                                  num_poses,
                                  num_atoms,
                                  block_size);
            });
        }
    }
}
