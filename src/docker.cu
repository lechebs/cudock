#include "hip/hip_runtime.h"
#include "docker.hpp"

#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm>
#include <type_traits>

#include "utils.cuh"
#include "swizzling.cuh"
#include "ligand.hpp"

#define WARP_SIZE 32
#define BLOCK_SIZE 128

namespace
{
    __constant__
    struct cuDock::Ligand::GPUData GPU_LIGAND_DATA;

    __constant__
    float * __restrict__ GPU_GMEM_VOXELS[cuDock::Pocket::NUM_CHANNELS];

    __constant__
    hipTextureObject_t GPU_TMEM_VOXELS[cuDock::Pocket::NUM_CHANNELS];

    __constant__ int NUM_CHANNELS;

    __constant__ int GRID_WIDTH;
    __constant__ int GRID_HEIGHT;
    __constant__ int GRID_DEPTH;
    __constant__ float GRID_CELL_SIZE;

    __constant__ int SWIZZLED_PADDED_WIDTH;
    __constant__ int SWIZZLED_PADDED_HEIGHT;
    __constant__ int SWIZZLED_TILE_SIZE_IN_BITS;

    __device__ __inline__ float warp_reduce(float value)
    {
        #pragma unroll 5
        for (int i = 0, d = 16; i < 5; ++i, d >>= 1) {
            value += __shfl_down_sync(0xffffffff, value, d);
        }

        return value;
    }

    __device__ float block_reduce(float value,
                                  int num_elements,
                                  float *shmem)
    {
        int lane_idx = threadIdx.x % WARP_SIZE;
        int warp_idx = threadIdx.x / WARP_SIZE;

        #pragma unroll
        while (num_elements > 1) {

            shmem[threadIdx.x] = 0;

            __syncthreads();

            if (warp_idx <= ((num_elements - 1) >> 5)) {
                value = warp_reduce(value);
                if (lane_idx == 0) {
                    shmem[warp_idx] = value;
                }
            }

            num_elements = ((num_elements - 1) >> 5) + 1;

            __syncthreads();

            if (warp_idx <= ((num_elements - 1) >> 5)) {
                value = shmem[threadIdx.x];
            }
        }

        return value;
    }

    __device__ __inline__ float3 _get_atom_pos(int idx)
    {
        float3 pos = {
            GPU_LIGAND_DATA.atoms_x[idx],
            GPU_LIGAND_DATA.atoms_y[idx],
            GPU_LIGAND_DATA.atoms_z[idx]
        };

        return pos;
    }

    __device__ __inline__ float score_pose_gmem(float3 pos,
                                                unsigned int mask)
    {
        float score = 0;

        int i = pos.x / GRID_CELL_SIZE;
        int j = pos.y / GRID_CELL_SIZE;
        int k = pos.z / GRID_CELL_SIZE;

        int grid_idx = k * GRID_WIDTH * GRID_HEIGHT + j * GRID_WIDTH + i;

        for (int c = 0; c < NUM_CHANNELS; ++c) {
            score += GPU_GMEM_VOXELS[c][grid_idx] * (mask & 1);
            mask >>= 1;
        }

        return score;
    }

    __device__ __inline__ float score_pose_gmem_lerp(float3 pos,
                                                     unsigned int mask)
    {
        float score = 0;
        /*
        int i = pos.x / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int j = pos.y / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int k = pos.z / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;

        int idxs[8];
        unsigned int dirs = 0b111110101100011010001000;
        #pragma unroll 8
        for (int d = 0; d < 8; ++d) {
            unsigned int dir = dirs & 7u;

            // Clamp to border
            int ni = max(0, min(i + (dir & 1u), GRID_WIDTH - 1));
            int nj = max(0, min(j + ((dir & 2u) > 1), GRID_HEIGHT - 1));
            int nk = max(0, min(k + ((dir & 4u) > 2), GRID_DEPTH - 1));

            idxs[d] = nk * GRID_WIDTH * GRID_HEIGHT +
                      nj * GRID_WIDTH + ni;

            dirs >>= 3;
        }

        for (int c = 0; c < NUM_CHANNELS; ++c) {
            for (int d = 0; d < 8; ++d) {
                score += GPU_GMEM_VOXELS[c][idxs[d]] * (mask & 1);
            }
            mask >>= 1;
        }
        */

        int lane_idx = threadIdx.x % 8;

        int i = pos.x / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int j = pos.y / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int k = pos.z / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;

        //int idxs[8];

        unsigned int dirs = 0b111110101100011010001000;
        unsigned dir = (dirs >> (3 * lane_idx)) & 7u;

        // Clamp to border
        int ni = max(0, min(i + (dir & 1u), GRID_WIDTH - 1));
        int nj = max(0, min(j + ((dir & 2u) > 1), GRID_HEIGHT - 1));
        int nk = max(0, min(k + ((dir & 4u) > 2), GRID_DEPTH - 1));

        int idx = nk * GRID_WIDTH * GRID_HEIGHT +
                  nj * GRID_WIDTH + ni;

        for (int c = 0; c < NUM_CHANNELS; ++c) {
            score += GPU_GMEM_VOXELS[c][idx] * (mask & 1);
            mask >>= 1;
        }

        score = warp_reduce(score);

        return score;
    }


    __device__ __inline__
    float score_pose_gmem_swizzled(int atom_idx,
                                   int num_atoms,
                                   int block_size,
                                   float3 pos,
                                   unsigned int mask)
    {
        float score = 0;

        /*

        NN_INTERPOLATE ---------

        int i = pos.x / GRID_CELL_SIZE;
        int j = pos.y / GRID_CELL_SIZE;
        int k = pos.z / GRID_CELL_SIZE;

        int idx = cuDock::Swizzling::
                  get_swizzled_idx(i,
                                   j,
                                   k,
                                   SWIZZLED_PADDED_WIDTH,
                                   SWIZZLED_PADDED_HEIGHT,
                                   SWIZZLED_TILE_SIZE_IN_BITS);

        for (int c = 0; c < NUM_CHANNELS; ++c) {
            score += GPU_GMEM_VOXELS[c][idx] * (mask & 1);
            mask >>= 1;
        }

        NN_INTERPOLATE -----------

        */

        int i = pos.x / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int j = pos.y / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int k = pos.z / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;

        for (int a = 0; a < WARP_SIZE; a += WARP_SIZE / 8) {

            int lane_idx = threadIdx.x % WARP_SIZE;
            int src_lane = i + lane_idx / 8;

            //WARNING: fix!
            //if (src_lane + threadIdx.x / WARP_SIZE * WARP_SIZE >= num_atoms) {
            //    continue;
            //}

            int ii = __shfl_sync(0xffffffff, i, src_lane);
            int jj = __shfl_sync(0xffffffff, j, src_lane);
            int kk = __shfl_sync(0xffffffff, k, src_lane);

            int dir_idx = threadIdx.x % 8;

            unsigned int dirs = 0b111110101100011010001000;
            unsigned dir = (dirs >> (3 * dir_idx)) & 7u;

            // Clamp to border
            int ni = max(0, min(ii + (dir & 1u), GRID_WIDTH - 1));
            int nj = max(0, min(jj + ((dir & 2u) > 1), GRID_HEIGHT - 1));
            int nk = max(0, min(kk + ((dir & 4u) > 2), GRID_DEPTH - 1));

            int idx = cuDock::Swizzling::
                      get_swizzled_idx(ni,
                                       nj,
                                       nk,
                                       SWIZZLED_PADDED_WIDTH,
                                       SWIZZLED_PADDED_HEIGHT,
                                       SWIZZLED_TILE_SIZE_IN_BITS);

            unsigned int m = mask;
            for (int c = 0; c < NUM_CHANNELS; ++c) {
                score += GPU_GMEM_VOXELS[c][idx] * (m & 1);
                m >>= 1;
            }
        }

        score = warp_reduce(score);

        /*

        LIN_INTERPOLATE ----------

        int i = pos.x / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int j = pos.y / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int k = pos.z / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;

        int idxs[8];
        unsigned int dirs = 0b111110101100011010001000;

        #pragma unroll 8
        for (int d = 0; d < 8; ++d) {
            unsigned int dir = dirs & 7u;

            // Clamp to border
            int ni = max(0, min(i + (dir & 1u), GRID_WIDTH - 1));
            int nj = max(0, min(j + ((dir & 2u) > 1), GRID_HEIGHT - 1));
            int nk = max(0, min(k + ((dir & 4u) > 2), GRID_DEPTH - 1));

            idxs[d] = cuDock::Swizzling::
                      get_swizzled_idx(ni,
                                       nj,
                                       nk,
                                       SWIZZLED_PADDED_WIDTH,
                                       SWIZZLED_PADDED_HEIGHT,
                                       SWIZZLED_TILE_SIZE_IN_BITS);

            dirs >>= 3;
        }

        for (int c = 0; c < NUM_CHANNELS; ++c) {
            for (int d = 0; d < 8; ++d) {
                score += GPU_GMEM_VOXELS[c][idxs[d]] * (mask & 1);
            }
            mask >>= 1;
        }

        LIN_INTERPOLATE ----------

        */


        /*
        LIN_INTERPOLATE with 8 threads per atom

        int i = pos.x / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int j = pos.y / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;
        int k = pos.z / GRID_CELL_SIZE - GRID_CELL_SIZE / 2;

        int dir_idx = threadIdx.x % 8;

        unsigned int dirs = 0b111110101100011010001000;
        unsigned dir = (dirs >> (3 * dir_idx)) & 7u;

        // Clamp to border
        int ni = max(0, min(i + (dir & 1u), GRID_WIDTH - 1));
        int nj = max(0, min(j + ((dir & 2u) > 1), GRID_HEIGHT - 1));
        int nk = max(0, min(k + ((dir & 4u) > 2), GRID_DEPTH - 1));

        int idx = cuDock::Swizzling::
                  get_swizzled_idx(ni,
                                   nj,
                                   nk,
                                   SWIZZLED_PADDED_WIDTH,
                                   SWIZZLED_PADDED_HEIGHT,
                                   SWIZZLED_TILE_SIZE_IN_BITS);


        for (int c = 0; c < NUM_CHANNELS; ++c) {
            score += GPU_GMEM_VOXELS[c][idx] * (mask & 1);
            mask >>= 1;
        }

        score = warp_reduce(score);
        */

        return score;
    }

    __device__ __inline__ float score_pose_tmem(float3 pos,
                                                unsigned int mask)
    {
        float score = 0;

        int tx = pos.x / GRID_CELL_SIZE;
        int ty = pos.y / GRID_CELL_SIZE;
        int tz = pos.z / GRID_CELL_SIZE;

        for (int c = 0; c < NUM_CHANNELS; ++c) {
            score += tex3D<float>(GPU_TMEM_VOXELS[c], tx, ty, tz) *
                     (mask & 1);
            mask >>= 1;
        }

        return score;
    }

    __global__ void score_poses(const float3 *translations,
                                const float3 *rotations,
                                float *scores,
                                int num_atoms,
                                int block_size,
                                enum GPUMemType mem_type,
                                enum InterpolateType int_type)
    {
        // Broadcasting to warp
        float3 t = translations[blockIdx.x];
        float3 r = rotations[blockIdx.x];

        float r_mat[9];
        cuDock::Docker::compute_rot_mat(r.x, r.y, r.z, r_mat);

        float score = 0.0;

        float3 pos;
        unsigned int mask;

        int idx = threadIdx.x;

        if (idx < num_atoms) {
            pos = _get_atom_pos(idx);
            cuDock::Docker::transform_atom_pos(pos.x,
                                               pos.y,
                                               pos.z,
                                               t.x,
                                               t.y,
                                               t.z,
                                               r_mat,
                                               pos.x,
                                               pos.y,
                                               pos.z);

            mask = GPU_LIGAND_DATA.atoms_channel_mask[idx];
        }

        if (mem_type == GPU_GMEM) {
            score += score_pose_gmem_lerp(pos, mask);

        } else if (mem_type == GPU_GMEM_SWIZZLED) {
            score += score_pose_gmem_swizzled(
                idx, num_atoms, block_size, pos, mask);

        } else if (mem_type == GPU_TMEM) {
            score += score_pose_tmem(pos, mask);
        }

        __syncthreads();

        //__shared__ float shmem[BLOCK_SIZE];
        //score = block_reduce(score, block_size, shmem);

        if (threadIdx.x == 0 && block_size > 0) {
            scores[blockIdx.x] = score;
        }
    }
}

namespace cuDock
{
     __device__ __host__ void Docker::compute_rot_mat(float rx,
                                                      float ry,
                                                      float rz,
                                                      float mat[])
    {
        float sinrx = sinf(rx);
        float cosrx = cosf(rx);
        float sinry = sinf(ry);
        float cosry = cosf(ry);
        float sinrz = sinf(rz);
        float cosrz = cosf(rz);

        mat[0] = cosry * cosrz;
        mat[1] = sinrx * sinry * cosrz - cosrx * sinrz;
        mat[2] = cosrx * sinry * cosrz + sinrx * sinrz;

        mat[3] = cosry * sinrz;
        mat[4] = sinrx * sinry * sinrz + cosrx * cosrz;
        mat[5] = cosrx * sinry * sinrz - sinrx * cosrz;

        mat[6] = -sinry;
        mat[7] = sinrx * cosry;
        mat[8] = cosrx * cosry;
    }

    __device__ __host__ void
    Docker::transform_atom_pos(float x,
                               float y,
                               float z,
                               float tx,
                               float ty,
                               float tz,
                               const float r_mat[],
                               float &x_dst,
                               float &y_dst,
                               float &z_dst)
    {
        x_dst = r_mat[0] * x + r_mat[1] * y + r_mat[2] * z + tx;
        y_dst = r_mat[3] * x + r_mat[4] * y + r_mat[5] * z + ty;
        z_dst = r_mat[6] * x + r_mat[7] * y + r_mat[8] * z + tz;
    }

    void Docker::to_gpu()
    {
        int num_poses = _translations.size();
        if (num_poses == 0) {
            return;
        }

        int alloc_size = num_poses * sizeof(float3);


        if (!_is_on_gpu) {
            _is_on_gpu = true;
            CUDA_CHECK_ERR(hipMalloc(&_gpu_translations, alloc_size));
            CUDA_CHECK_ERR(hipMalloc(&_gpu_rotations, alloc_size));
            CUDA_CHECK_ERR(hipMalloc(&_gpu_scores,
                                      num_poses * sizeof(float)));
        }

        std::vector<float3> t_buffer;
        std::vector<float3> r_buffer;
        t_buffer.reserve(num_poses);
        r_buffer.reserve(num_poses);

        for (int i = 0; i < num_poses; ++i) {
            t_buffer[i].x = _translations[i][0];
            t_buffer[i].y = _translations[i][1];
            t_buffer[i].z = _translations[i][2];
            r_buffer[i].x = _rotations[i][0];
            r_buffer[i].y = _rotations[i][1];
            r_buffer[i].z = _rotations[i][2];
        }

        // Copying poses to global memory

        CUDA_CHECK_ERR(hipMemcpy(_gpu_translations,
                                  t_buffer.data(),
                                  alloc_size,
                                  hipMemcpyHostToDevice));

        CUDA_CHECK_ERR(hipMemcpy(_gpu_rotations,
                                  r_buffer.data(),
                                  alloc_size,
                                  hipMemcpyHostToDevice));

        // Copying ligand data to SoA constant memory buffer

        const std::vector<Ligand::Atom> &atoms = _ligand.get_atoms();
        int num_atoms = atoms.size();

        struct Ligand::GPUData ligand_data;
        for (int i = 0; i < num_atoms; ++i) {
            const Ligand::Atom &atom = atoms[i];
            ligand_data.atoms_x[i] = atom.pos[0];
            ligand_data.atoms_y[i] = atom.pos[1];
            ligand_data.atoms_z[i] = atom.pos[2];
            ligand_data.atoms_mass[i] = Ligand::get_atom_mass(atom.type);
            // ligand_data.atom_type[i] = atom.type;
            ligand_data.atoms_channel_mask[i] =
                Ligand::get_atom_channel_mask(atom.type);
        }

        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(GPU_LIGAND_DATA),
                                          &ligand_data,
                                          sizeof(struct Ligand::GPUData)));

        // Copying various data to constant memory

        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(NUM_CHANNELS),
                                          &Pocket::NUM_CHANNELS,
                                          sizeof(int)));

        int shape = _pocket.get_shape(0);
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(GRID_WIDTH), &shape, sizeof(int)));

        shape = _pocket.get_shape(1);
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(GRID_HEIGHT), &shape, sizeof(int)));

        shape = _pocket.get_shape(2);
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(GRID_DEPTH), &shape, sizeof(int)));

        float cell_size = _pocket.get_cell_size();
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(GRID_CELL_SIZE),
                                          &cell_size,
                                          sizeof(float)));

        int tile_size_in_bits = 4;
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(SWIZZLED_TILE_SIZE_IN_BITS),
                                          &tile_size_in_bits,
                                          sizeof(int)));

        int padded_size = _pocket.get_shape(0) +
                          cuDock::Swizzling::
                          get_padding_size(_pocket.get_shape(0),
                                           tile_size_in_bits);
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(SWIZZLED_PADDED_WIDTH),
                                          &padded_size,
                                          sizeof(int)));

        padded_size = _pocket.get_shape(1) +
                      cuDock::Swizzling::
                      get_padding_size(_pocket.get_shape(1),
                                       tile_size_in_bits);
        CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(SWIZZLED_PADDED_HEIGHT),
                                          &padded_size,
                                          sizeof(int)));
    }

    void Docker::off_gpu()
    {
        if (_is_on_gpu) {
            _is_on_gpu = false;
            CUDA_CHECK_ERR(hipFree(_gpu_translations));
            CUDA_CHECK_ERR(hipFree(_gpu_rotations));
            CUDA_CHECK_ERR(hipFree(_gpu_scores));
        }
    }

    void Docker::get_scores(std::vector<float> &dst) const
    {
        dst.resize(_translations.size());

        if (_is_on_gpu) {
            hipMemcpy(dst.data(),
                       _gpu_scores,
                       dst.size() * sizeof(float),
                       hipMemcpyDeviceToHost);
        } else {
            std::copy(_scores.begin(), _scores.end(), dst.begin());
        }
    }

    void Docker::_score_poses_gpu(int num_poses)
    {
        if (_pocket.is_on_gpu(GPU_GMEM) ||
            _pocket.is_on_gpu(GPU_GMEM_SWIZZLED)) {
            // Copy pocket voxels gmem pointers to constant memory
            CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(
                GPU_GMEM_VOXELS),
                _pocket.get_gpu_gmem_voxels().data(),
                sizeof(float *) * Pocket::NUM_CHANNELS));
        } else {
            // Copy pocket voxels tmem pointers to constant memory
            CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(
                GPU_TMEM_VOXELS),
                _pocket.get_gpu_tmem_voxels().data(),
                sizeof(hipTextureObject_t) * Pocket::NUM_CHANNELS));
        }

        int num_atoms = _ligand.get_num_atoms();
        int block_size = std::ceil((float) num_atoms / WARP_SIZE) * WARP_SIZE;

        if (_pocket.is_on_gpu(GPU_GMEM)) {
            CUDA_TIME_EXEC("_score_gmem", [&](){
                score_poses<<<
                    num_poses,
                    block_size>>>(_gpu_translations,
                                  _gpu_rotations,
                                  _gpu_scores,
                                  num_atoms,
                                  block_size,
                                  GPU_GMEM,
                                  _pocket.get_interpolate());
            });
        } else if (_pocket.is_on_gpu(GPU_GMEM_SWIZZLED)) {
            CUDA_TIME_EXEC("_score_gmem_swizzled", [&](){
                score_poses<<<
                    num_poses,
                    block_size>>>(_gpu_translations,
                                  _gpu_rotations,
                                  _gpu_scores,
                                  num_atoms,
                                  block_size,
                                  GPU_GMEM_SWIZZLED,
                                  _pocket.get_interpolate());
            });
        } else {
            CUDA_TIME_EXEC("_score_tmem", [&](){
                score_poses<<<
                    num_poses,
                    block_size>>>(_gpu_translations,
                                  _gpu_rotations,
                                  _gpu_scores,
                                  num_atoms,
                                  block_size,
                                  GPU_TMEM,
                                  _pocket.get_interpolate());
            });

        }
    }
}
